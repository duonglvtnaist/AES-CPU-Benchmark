#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include "aes.h"

#define AES_BLOCK_SIZE 16

__device__ void AES_init_ctx_iv_device(struct AES_ctx* ctx, const uint8_t* key, const uint8_t* iv) {
    // Initialize the AES context with the key
    memcpy(ctx->RoundKey, key, AES_BLOCK_SIZE);
    memcpy(ctx->Iv, iv, AES_BLOCK_SIZE);
}

__device__ void AES_ECB_encrypt_device(const struct AES_ctx* ctx, uint8_t* buf) {
    // Implement the AES ECB encryption logic here
    // This is a simplified example and may not be secure
    for (int i = 0; i < AES_BLOCK_SIZE; ++i) {
        buf[i] ^= ctx->RoundKey[i];
    }
}

__device__ void AES_CBC_encrypt_buffer_device(struct AES_ctx* ctx, uint8_t* buf, size_t length) {
    uint8_t* Iv = ctx->Iv;
    for (size_t i = 0; i < length; i += AES_BLOCK_SIZE) {
        for (size_t j = 0; j < AES_BLOCK_SIZE; ++j) {
            buf[i + j] ^= Iv[j];
        }
        AES_ECB_encrypt_device(ctx, buf + i);
        memcpy(Iv, buf + i, AES_BLOCK_SIZE);
    }
}

__global__ void aes_encrypt_cbc_kernel(uint8_t* d_in, uint8_t* d_out, uint8_t* d_key, uint8_t* d_iv, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length / AES_BLOCK_SIZE) {
        struct AES_ctx ctx;
        AES_init_ctx_iv_device(&ctx, d_key, d_iv);
        AES_CBC_encrypt_buffer_device(&ctx, d_in + idx * AES_BLOCK_SIZE, AES_BLOCK_SIZE);
        memcpy(d_out + idx * AES_BLOCK_SIZE, d_in + idx * AES_BLOCK_SIZE, AES_BLOCK_SIZE);
    }
}

void aes_encrypt_cbc(uint8_t* in, uint8_t* out, uint8_t* key, uint8_t* iv, int length) {
    uint8_t *d_in, *d_out, *d_key, *d_iv;

    hipMalloc((void**)&d_in, length);
    hipMalloc((void**)&d_out, length);
    hipMalloc((void**)&d_key, AES_BLOCK_SIZE);
    hipMalloc((void**)&d_iv, AES_BLOCK_SIZE);

    hipMemcpy(d_in, in, length, hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, AES_BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_iv, iv, AES_BLOCK_SIZE, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (length / AES_BLOCK_SIZE + blockSize - 1) / blockSize;
    aes_encrypt_cbc_kernel<<<numBlocks, blockSize>>>(d_in, d_out, d_key, d_iv, length);

    hipMemcpy(out, d_out, length, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_key);
    hipFree(d_iv);
}

int main() {
#if defined(AES256)
    uint8_t key[] = { 0x60, 0x3d, 0xeb, 0x10, 0x15, 0xca, 0x71, 0xbe, 0x2b, 0x73, 0xae, 0xf0, 0x85, 0x7d, 0x77, 0x81,
                      0x1f, 0x35, 0x2c, 0x07, 0x3b, 0x61, 0x08, 0xd7, 0x2d, 0x98, 0x10, 0xa3, 0x09, 0x14, 0xdf, 0xf4 };
#elif defined(AES192)
    uint8_t key[] = { 0x8e, 0x73, 0xb0, 0xf7, 0xda, 0x0e, 0x64, 0x52, 0xc8, 0x10, 0xf3, 0x2b, 0x80, 0x90, 0x79, 0xe5, 0x62, 0xf8, 0xea, 0xd2, 0x52, 0x2c, 0x6b, 0x7b };
#elif defined(AES128)
    uint8_t key[] = { 0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6, 0xab, 0xf7, 0x15, 0x88, 0x09, 0xcf, 0x4f, 0x3c };
#endif
    uint8_t iv[]  = { 0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f };
    uint8_t in[]  = { 0x6b, 0xc1, 0xbe, 0xe2, 0x2e, 0x40, 0x9f, 0x96, 0xe9, 0x3d, 0x7e, 0x11, 0x73, 0x93, 0x17, 0x2a,
                      0xae, 0x2d, 0x8a, 0x57, 0x1e, 0x03, 0xac, 0x9c, 0x9e, 0xb7, 0x6f, 0xac, 0x45, 0xaf, 0x8e, 0x51,
                      0x30, 0xc8, 0x1c, 0x46, 0xa3, 0x5c, 0xe4, 0x11, 0xe5, 0xfb, 0xc1, 0x19, 0x1a, 0x0a, 0x52, 0xef,
                      0xf6, 0x9f, 0x24, 0x45, 0xdf, 0x4f, 0x9b, 0x17, 0xad, 0x2b, 0x41, 0x7b, 0xe6, 0x6c, 0x37, 0x10 };
    uint8_t out[sizeof(in)];

    aes_encrypt_cbc(in, out, key, iv, sizeof(in));

    printf("CBC encrypt: ");
    if (0 == memcmp((char*) out, (char*) in, sizeof(in))) {
        printf("SUCCESS!\n");
    } else {
        printf("FAILURE!\n");
    }

    return 0;
}